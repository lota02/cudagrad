#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void vecAdd(float* A, float* B, float* C) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  C[i] = A[i] + B[i];
}

__global__ void vecSub(float* A, float* B, float* C) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  C[i] = A[i] - B[i];
}

__global__ void vecMul(float* A, float* B, float* C) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  C[i] = A[i] * B[i];
}

__global__ void vecDiv(float* A, float* B, float* C) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  C[i] = A[i] / B[i];
}

int main(void) {
  int N = 1024;
  size_t size = N * sizeof(float);

  float* h_A = (float*)malloc(size);
  float* h_B = (float*)malloc(size);
  float* h_C = (float*)malloc(size);

  for(int i = 0; i < N; ++i) {
    h_A[i] = i;
    h_B[i] = i;
  }

  float *d_A, *d_B, *d_C;
  hipMalloc(&d_A, size);
  hipMalloc(&d_B, size);
  hipMalloc(&d_C, size);

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  vecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C);

  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  for(int i = 0; i < N; ++i) {
    if (h_A[i] + h_B[i] != h_C[i]) {
      printf("Results do not match at index %d!\n", i);
      break;
    }
  }

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  free(h_A);
  free(h_B);
  free(h_C);

  return 0;
}


#include <hip/hip_runtime.h>
#include <stdio.h>

namespace cg {

__global__ void helloFromGPU() { printf("Hello, GPU!\n"); }

extern "C" void hello() {
  helloFromGPU<<<1, 1>>>();
  hipDeviceSynchronize();
}

} // namespace cg
